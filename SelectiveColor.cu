#include "hip/hip_runtime.h"
#include "SelectiveColor.h"

#include "opencv2/cudev/util/saturate_cast.hpp"

#include <hip/hip_runtime_api.h>

#include <math_functions.h>
#include <hip/device_functions.h>

static __constant__ float one_lut[256] = { 1.0f };
static __constant__ float min_lut[256] = {
    -0.0f / 255.0f, -1.0f / 255.0f, -2.0f / 255.0f, -3.0f / 255.0f, -4.0f / 255.0f, -5.0f / 255.0f, -6.0f / 255.0f, -7.0f / 255.0f,
    -8.0f / 255.0f, -9.0f / 255.0f, -10.0f / 255.0f, -11.0f / 255.0f, -12.0f / 255.0f, -13.0f / 255.0f, -14.0f / 255.0f, -15.0f / 255.0f,
    -16.0f / 255.0f, -17.0f / 255.0f, -18.0f / 255.0f, -19.0f / 255.0f, -20.0f / 255.0f, -21.0f / 255.0f, -22.0f / 255.0f, -23.0f / 255.0f,
    -24.0f / 255.0f, -25.0f / 255.0f, -26.0f / 255.0f, -27.0f / 255.0f, -28.0f / 255.0f, -29.0f / 255.0f, -30.0f / 255.0f, -31.0f / 255.0f,
    -32.0f / 255.0f, -33.0f / 255.0f, -34.0f / 255.0f, -35.0f / 255.0f, -36.0f / 255.0f, -37.0f / 255.0f, -38.0f / 255.0f, -39.0f / 255.0f,
    -40.0f / 255.0f, -41.0f / 255.0f, -42.0f / 255.0f, -43.0f / 255.0f, -44.0f / 255.0f, -45.0f / 255.0f, -46.0f / 255.0f, -47.0f / 255.0f,
    -48.0f / 255.0f, -49.0f / 255.0f, -50.0f / 255.0f, -51.0f / 255.0f, -52.0f / 255.0f, -53.0f / 255.0f, -54.0f / 255.0f, -55.0f / 255.0f,
    -56.0f / 255.0f, -57.0f / 255.0f, -58.0f / 255.0f, -59.0f / 255.0f, -60.0f / 255.0f, -61.0f / 255.0f, -62.0f / 255.0f, -63.0f / 255.0f,
    -64.0f / 255.0f, -65.0f / 255.0f, -66.0f / 255.0f, -67.0f / 255.0f, -68.0f / 255.0f, -69.0f / 255.0f, -70.0f / 255.0f, -71.0f / 255.0f,
    -72.0f / 255.0f, -73.0f / 255.0f, -74.0f / 255.0f, -75.0f / 255.0f, -76.0f / 255.0f, -77.0f / 255.0f, -78.0f / 255.0f, -79.0f / 255.0f,
    -80.0f / 255.0f, -81.0f / 255.0f, -82.0f / 255.0f, -83.0f / 255.0f, -84.0f / 255.0f, -85.0f / 255.0f, -86.0f / 255.0f, -87.0f / 255.0f,
    -88.0f / 255.0f, -89.0f / 255.0f, -90.0f / 255.0f, -91.0f / 255.0f, -92.0f / 255.0f, -93.0f / 255.0f, -94.0f / 255.0f, -95.0f / 255.0f,
    -96.0f / 255.0f, -97.0f / 255.0f, -98.0f / 255.0f, -99.0f / 255.0f, -100.0f / 255.0f, -101.0f / 255.0f, -102.0f / 255.0f, -103.0f / 255.0f,
    -104.0f / 255.0f, -105.0f / 255.0f, -106.0f / 255.0f, -107.0f / 255.0f, -108.0f / 255.0f, -109.0f / 255.0f, -110.0f / 255.0f, -111.0f / 255.0f,
    -112.0f / 255.0f, -113.0f / 255.0f, -114.0f / 255.0f, -115.0f / 255.0f, -116.0f / 255.0f, -117.0f / 255.0f, -118.0f / 255.0f, -119.0f / 255.0f,
    -120.0f / 255.0f, -121.0f / 255.0f, -122.0f / 255.0f, -123.0f / 255.0f, -124.0f / 255.0f, -125.0f / 255.0f, -126.0f / 255.0f, -127.0f / 255.0f,
    -128.0f / 255.0f, -129.0f / 255.0f, -130.0f / 255.0f, -131.0f / 255.0f, -132.0f / 255.0f, -133.0f / 255.0f, -134.0f / 255.0f, -135.0f / 255.0f,
    -136.0f / 255.0f, -137.0f / 255.0f, -138.0f / 255.0f, -139.0f / 255.0f, -140.0f / 255.0f, -141.0f / 255.0f, -142.0f / 255.0f, -143.0f / 255.0f,
    -144.0f / 255.0f, -145.0f / 255.0f, -146.0f / 255.0f, -147.0f / 255.0f, -148.0f / 255.0f, -149.0f / 255.0f, -150.0f / 255.0f, -151.0f / 255.0f,
    -152.0f / 255.0f, -153.0f / 255.0f, -154.0f / 255.0f, -155.0f / 255.0f, -156.0f / 255.0f, -157.0f / 255.0f, -158.0f / 255.0f, -159.0f / 255.0f,
    -160.0f / 255.0f, -161.0f / 255.0f, -162.0f / 255.0f, -163.0f / 255.0f, -164.0f / 255.0f, -165.0f / 255.0f, -166.0f / 255.0f, -167.0f / 255.0f,
    -168.0f / 255.0f, -169.0f / 255.0f, -170.0f / 255.0f, -171.0f / 255.0f, -172.0f / 255.0f, -173.0f / 255.0f, -174.0f / 255.0f, -175.0f / 255.0f,
    -176.0f / 255.0f, -177.0f / 255.0f, -178.0f / 255.0f, -179.0f / 255.0f, -180.0f / 255.0f, -181.0f / 255.0f, -182.0f / 255.0f, -183.0f / 255.0f,
    -184.0f / 255.0f, -185.0f / 255.0f, -186.0f / 255.0f, -187.0f / 255.0f, -188.0f / 255.0f, -189.0f / 255.0f, -190.0f / 255.0f, -191.0f / 255.0f,
    -192.0f / 255.0f, -193.0f / 255.0f, -194.0f / 255.0f, -195.0f / 255.0f, -196.0f / 255.0f, -197.0f / 255.0f, -198.0f / 255.0f, -199.0f / 255.0f,
    -200.0f / 255.0f, -201.0f / 255.0f, -202.0f / 255.0f, -203.0f / 255.0f, -204.0f / 255.0f, -205.0f / 255.0f, -206.0f / 255.0f, -207.0f / 255.0f,
    -208.0f / 255.0f, -209.0f / 255.0f, -210.0f / 255.0f, -211.0f / 255.0f, -212.0f / 255.0f, -213.0f / 255.0f, -214.0f / 255.0f, -215.0f / 255.0f,
    -216.0f / 255.0f, -217.0f / 255.0f, -218.0f / 255.0f, -219.0f / 255.0f, -220.0f / 255.0f, -221.0f / 255.0f, -222.0f / 255.0f, -223.0f / 255.0f,
    -224.0f / 255.0f, -225.0f / 255.0f, -226.0f / 255.0f, -227.0f / 255.0f, -228.0f / 255.0f, -229.0f / 255.0f, -230.0f / 255.0f, -231.0f / 255.0f,
    -232.0f / 255.0f, -233.0f / 255.0f, -234.0f / 255.0f, -235.0f / 255.0f, -236.0f / 255.0f, -237.0f / 255.0f, -238.0f / 255.0f, -239.0f / 255.0f,
    -240.0f / 255.0f, -241.0f / 255.0f, -242.0f / 255.0f, -243.0f / 255.0f, -244.0f / 255.0f, -245.0f / 255.0f, -246.0f / 255.0f, -247.0f / 255.0f,
    -248.0f / 255.0f, -249.0f / 255.0f, -250.0f / 255.0f, -251.0f / 255.0f, -252.0f / 255.0f, -253.0f / 255.0f, -254.0f / 255.0f, -255.0f / 255.0f
};

static __constant__ float max_lut[256] = {
    1.0f - 0.0f / 255.0f, 1.0f - 1.0f / 255.0f, 1.0f - 2.0f / 255.0f, 1.0f - 3.0f / 255.0f, 1.0f - 4.0f / 255.0f, 1.0f - 5.0f / 255.0f, 1.0f - 6.0f / 255.0f, 1.0f - 7.0f / 255.0f,
    1.0f - 8.0f / 255.0f, 1.0f - 9.0f / 255.0f, 1.0f - 10.0f / 255.0f, 1.0f - 11.0f / 255.0f, 1.0f - 12.0f / 255.0f, 1.0f - 13.0f / 255.0f, 1.0f - 14.0f / 255.0f, 1.0f - 15.0f / 255.0f,
    1.0f - 16.0f / 255.0f, 1.0f - 17.0f / 255.0f, 1.0f - 18.0f / 255.0f, 1.0f - 19.0f / 255.0f, 1.0f - 20.0f / 255.0f, 1.0f - 21.0f / 255.0f, 1.0f - 22.0f / 255.0f, 1.0f - 23.0f / 255.0f,
    1.0f - 24.0f / 255.0f, 1.0f - 25.0f / 255.0f, 1.0f - 26.0f / 255.0f, 1.0f - 27.0f / 255.0f, 1.0f - 28.0f / 255.0f, 1.0f - 29.0f / 255.0f, 1.0f - 30.0f / 255.0f, 1.0f - 31.0f / 255.0f,
    1.0f - 32.0f / 255.0f, 1.0f - 33.0f / 255.0f, 1.0f - 34.0f / 255.0f, 1.0f - 35.0f / 255.0f, 1.0f - 36.0f / 255.0f, 1.0f - 37.0f / 255.0f, 1.0f - 38.0f / 255.0f, 1.0f - 39.0f / 255.0f,
    1.0f - 40.0f / 255.0f, 1.0f - 41.0f / 255.0f, 1.0f - 42.0f / 255.0f, 1.0f - 43.0f / 255.0f, 1.0f - 44.0f / 255.0f, 1.0f - 45.0f / 255.0f, 1.0f - 46.0f / 255.0f, 1.0f - 47.0f / 255.0f,
    1.0f - 48.0f / 255.0f, 1.0f - 49.0f / 255.0f, 1.0f - 50.0f / 255.0f, 1.0f - 51.0f / 255.0f, 1.0f - 52.0f / 255.0f, 1.0f - 53.0f / 255.0f, 1.0f - 54.0f / 255.0f, 1.0f - 55.0f / 255.0f,
    1.0f - 56.0f / 255.0f, 1.0f - 57.0f / 255.0f, 1.0f - 58.0f / 255.0f, 1.0f - 59.0f / 255.0f, 1.0f - 60.0f / 255.0f, 1.0f - 61.0f / 255.0f, 1.0f - 62.0f / 255.0f, 1.0f - 63.0f / 255.0f,
    1.0f - 64.0f / 255.0f, 1.0f - 65.0f / 255.0f, 1.0f - 66.0f / 255.0f, 1.0f - 67.0f / 255.0f, 1.0f - 68.0f / 255.0f, 1.0f - 69.0f / 255.0f, 1.0f - 70.0f / 255.0f, 1.0f - 71.0f / 255.0f,
    1.0f - 72.0f / 255.0f, 1.0f - 73.0f / 255.0f, 1.0f - 74.0f / 255.0f, 1.0f - 75.0f / 255.0f, 1.0f - 76.0f / 255.0f, 1.0f - 77.0f / 255.0f, 1.0f - 78.0f / 255.0f, 1.0f - 79.0f / 255.0f,
    1.0f - 80.0f / 255.0f, 1.0f - 81.0f / 255.0f, 1.0f - 82.0f / 255.0f, 1.0f - 83.0f / 255.0f, 1.0f - 84.0f / 255.0f, 1.0f - 85.0f / 255.0f, 1.0f - 86.0f / 255.0f, 1.0f - 87.0f / 255.0f,
    1.0f - 88.0f / 255.0f, 1.0f - 89.0f / 255.0f, 1.0f - 90.0f / 255.0f, 1.0f - 91.0f / 255.0f, 1.0f - 92.0f / 255.0f, 1.0f - 93.0f / 255.0f, 1.0f - 94.0f / 255.0f, 1.0f - 95.0f / 255.0f,
    1.0f - 96.0f / 255.0f, 1.0f - 97.0f / 255.0f, 1.0f - 98.0f / 255.0f, 1.0f - 99.0f / 255.0f, 1.0f - 100.0f / 255.0f, 1.0f - 101.0f / 255.0f, 1.0f - 102.0f / 255.0f, 1.0f - 103.0f / 255.0f,
    1.0f - 104.0f / 255.0f, 1.0f - 105.0f / 255.0f, 1.0f - 106.0f / 255.0f, 1.0f - 107.0f / 255.0f, 1.0f - 108.0f / 255.0f, 1.0f - 109.0f / 255.0f, 1.0f - 110.0f / 255.0f, 1.0f - 111.0f / 255.0f,
    1.0f - 112.0f / 255.0f, 1.0f - 113.0f / 255.0f, 1.0f - 114.0f / 255.0f, 1.0f - 115.0f / 255.0f, 1.0f - 116.0f / 255.0f, 1.0f - 117.0f / 255.0f, 1.0f - 118.0f / 255.0f, 1.0f - 119.0f / 255.0f,
    1.0f - 120.0f / 255.0f, 1.0f - 121.0f / 255.0f, 1.0f - 122.0f / 255.0f, 1.0f - 123.0f / 255.0f, 1.0f - 124.0f / 255.0f, 1.0f - 125.0f / 255.0f, 1.0f - 126.0f / 255.0f, 1.0f - 127.0f / 255.0f,
    1.0f - 128.0f / 255.0f, 1.0f - 129.0f / 255.0f, 1.0f - 130.0f / 255.0f, 1.0f - 131.0f / 255.0f, 1.0f - 132.0f / 255.0f, 1.0f - 133.0f / 255.0f, 1.0f - 134.0f / 255.0f, 1.0f - 135.0f / 255.0f,
    1.0f - 136.0f / 255.0f, 1.0f - 137.0f / 255.0f, 1.0f - 138.0f / 255.0f, 1.0f - 139.0f / 255.0f, 1.0f - 140.0f / 255.0f, 1.0f - 141.0f / 255.0f, 1.0f - 142.0f / 255.0f, 1.0f - 143.0f / 255.0f,
    1.0f - 144.0f / 255.0f, 1.0f - 145.0f / 255.0f, 1.0f - 146.0f / 255.0f, 1.0f - 147.0f / 255.0f, 1.0f - 148.0f / 255.0f, 1.0f - 149.0f / 255.0f, 1.0f - 150.0f / 255.0f, 1.0f - 151.0f / 255.0f,
    1.0f - 152.0f / 255.0f, 1.0f - 153.0f / 255.0f, 1.0f - 154.0f / 255.0f, 1.0f - 155.0f / 255.0f, 1.0f - 156.0f / 255.0f, 1.0f - 157.0f / 255.0f, 1.0f - 158.0f / 255.0f, 1.0f - 159.0f / 255.0f,
    1.0f - 160.0f / 255.0f, 1.0f - 161.0f / 255.0f, 1.0f - 162.0f / 255.0f, 1.0f - 163.0f / 255.0f, 1.0f - 164.0f / 255.0f, 1.0f - 165.0f / 255.0f, 1.0f - 166.0f / 255.0f, 1.0f - 167.0f / 255.0f,
    1.0f - 168.0f / 255.0f, 1.0f - 169.0f / 255.0f, 1.0f - 170.0f / 255.0f, 1.0f - 171.0f / 255.0f, 1.0f - 172.0f / 255.0f, 1.0f - 173.0f / 255.0f, 1.0f - 174.0f / 255.0f, 1.0f - 175.0f / 255.0f,
    1.0f - 176.0f / 255.0f, 1.0f - 177.0f / 255.0f, 1.0f - 178.0f / 255.0f, 1.0f - 179.0f / 255.0f, 1.0f - 180.0f / 255.0f, 1.0f - 181.0f / 255.0f, 1.0f - 182.0f / 255.0f, 1.0f - 183.0f / 255.0f,
    1.0f - 184.0f / 255.0f, 1.0f - 185.0f / 255.0f, 1.0f - 186.0f / 255.0f, 1.0f - 187.0f / 255.0f, 1.0f - 188.0f / 255.0f, 1.0f - 189.0f / 255.0f, 1.0f - 190.0f / 255.0f, 1.0f - 191.0f / 255.0f,
    1.0f - 192.0f / 255.0f, 1.0f - 193.0f / 255.0f, 1.0f - 194.0f / 255.0f, 1.0f - 195.0f / 255.0f, 1.0f - 196.0f / 255.0f, 1.0f - 197.0f / 255.0f, 1.0f - 198.0f / 255.0f, 1.0f - 199.0f / 255.0f,
    1.0f - 200.0f / 255.0f, 1.0f - 201.0f / 255.0f, 1.0f - 202.0f / 255.0f, 1.0f - 203.0f / 255.0f, 1.0f - 204.0f / 255.0f, 1.0f - 205.0f / 255.0f, 1.0f - 206.0f / 255.0f, 1.0f - 207.0f / 255.0f,
    1.0f - 208.0f / 255.0f, 1.0f - 209.0f / 255.0f, 1.0f - 210.0f / 255.0f, 1.0f - 211.0f / 255.0f, 1.0f - 212.0f / 255.0f, 1.0f - 213.0f / 255.0f, 1.0f - 214.0f / 255.0f, 1.0f - 215.0f / 255.0f,
    1.0f - 216.0f / 255.0f, 1.0f - 217.0f / 255.0f, 1.0f - 218.0f / 255.0f, 1.0f - 219.0f / 255.0f, 1.0f - 220.0f / 255.0f, 1.0f - 221.0f / 255.0f, 1.0f - 222.0f / 255.0f, 1.0f - 223.0f / 255.0f,
    1.0f - 224.0f / 255.0f, 1.0f - 225.0f / 255.0f, 1.0f - 226.0f / 255.0f, 1.0f - 227.0f / 255.0f, 1.0f - 228.0f / 255.0f, 1.0f - 229.0f / 255.0f, 1.0f - 230.0f / 255.0f, 1.0f - 231.0f / 255.0f,
    1.0f - 232.0f / 255.0f, 1.0f - 233.0f / 255.0f, 1.0f - 234.0f / 255.0f, 1.0f - 235.0f / 255.0f, 1.0f - 236.0f / 255.0f, 1.0f - 237.0f / 255.0f, 1.0f - 238.0f / 255.0f, 1.0f - 239.0f / 255.0f,
    1.0f - 240.0f / 255.0f, 1.0f - 241.0f / 255.0f, 1.0f - 242.0f / 255.0f, 1.0f - 243.0f / 255.0f, 1.0f - 244.0f / 255.0f, 1.0f - 245.0f / 255.0f, 1.0f - 246.0f / 255.0f, 1.0f - 247.0f / 255.0f,
    1.0f - 248.0f / 255.0f, 1.0f - 249.0f / 255.0f, 1.0f - 250.0f / 255.0f, 1.0f - 251.0f / 255.0f, 1.0f - 252.0f / 255.0f, 1.0f - 253.0f / 255.0f, 1.0f - 254.0f / 255.0f, 1.0f - 255.0f / 255.0f
};

static __device__ void min_med_max(const unsigned char* color, unsigned char& min, unsigned char& med, unsigned char& max)
{
    unsigned char temp = 0;
    min = color[0], med = color[1], max = color[2];
    if (min > med)
    {
        temp = min;
        min = med;
        med = temp;
    }
    if (med > max)
    {
        temp = med;
        med = max;
        max = temp;
    }
    if (min > med)
    {
        temp = min;
        min = med;
        med = temp;
    }
}

static __device__ float adjust_component(int mode, unsigned char component_color, float omega, float adjust, float black)
{
    static float* mode_lut[2] = { one_lut, max_lut };
    
    (void)black;

    float result = (/*(-1.0f - adjust) * black*/0.0f - adjust) * mode_lut[mode][component_color];
    if (result < min_lut[component_color])
    {
        result = min_lut[component_color];
    }
    else if (result > max_lut[component_color])
    {
        result = max_lut[component_color];
    }

    return result * omega;
}

static __global__ void cuda_adjust_color_kernel(unsigned char* dst, int dst_step, const unsigned char* src, int src_step, int width, int height, int channels, int mode, int component, float cyan, float magtenta, float yellow, float black)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        unsigned char* dst_color = dst + y * dst_step + x * channels;
        const unsigned char* src_color = src + y * src_step + x * channels;

        unsigned char min = 0, med = 0, max = 0;
        min_med_max(src_color, min, med, max);

        bool valid = false;
        float result[3] = { 0.0f };
        float omega = 0.0f;
        switch (component)
        {
        case SelectiveColor::Component_Red:
        case SelectiveColor::Component_Green:
        case SelectiveColor::Component_Blue:
        {
            valid = (max == src_color[component]);
            omega = static_cast<float>(max) - static_cast<float>(med);
            break;
        }
        case SelectiveColor::Component_Cyan:
        case SelectiveColor::Component_Magtenta:
        case SelectiveColor::Component_Yellow:
        {
            valid = (min == src_color[component - 3]);
            omega = static_cast<float>(med) - static_cast<float>(min);
            break;
        }
        default:
            break;
        }

        if (valid)
        {
            result[SelectiveColor::Component_Red] = adjust_component(mode, src_color[SelectiveColor::Component_Red], omega, cyan, black);
            result[SelectiveColor::Component_Green] = adjust_component(mode, src_color[SelectiveColor::Component_Green], omega, magtenta, black);
            result[SelectiveColor::Component_Blue] = adjust_component(mode, src_color[SelectiveColor::Component_Blue], omega, yellow, black);
        }

        dst_color[SelectiveColor::Component_Red] = cv::cudev::saturate_cast<unsigned char>(src_color[SelectiveColor::Component_Red] + result[SelectiveColor::Component_Red]);
        dst_color[SelectiveColor::Component_Green] = cv::cudev::saturate_cast<unsigned char>(src_color[SelectiveColor::Component_Green] + result[SelectiveColor::Component_Green]);
        dst_color[SelectiveColor::Component_Blue] = cv::cudev::saturate_cast<unsigned char>(src_color[SelectiveColor::Component_Blue] + result[SelectiveColor::Component_Blue]);
    }
}

int cuda_adjust_color(cv::cuda::GpuMat& dst, const cv::cuda::GpuMat& src, int mode, int component, float cyan, float magtenta, float yellow, cv::cuda::Stream& stream)
{
    dim3 block(32, 8, 1);
    dim3 grid((dst.cols + 31) / block.x, (dst.rows + 7) / block.y, 1);

    cuda_adjust_color_kernel << < grid, block, 0, static_cast<hipStream_t>(stream.cudaPtr()) >> > (
        (unsigned char*)(dst.data), (int)(dst.step), 
        (const unsigned char*)(src.data), (int)(src.step),
        src.cols, src.rows, src.channels(), 
        mode, component, cyan, magtenta, yellow, 0.0f);

    return static_cast<int>(hipPeekAtLastError());
}

